#include "hip/hip_runtime.h"
/***************************************************************************
 *   Copyright (C) 2008 by Benjamin Knispel, Holger Pletsch                *
 *   benjamin.knispel[AT]aei.mpg.de                                        *
 *   Copyright (C) 2009,2010 by Oliver Bock                                *
 *   oliver.bock[AT]aei.mpg.de                                             *
 *   Copyright (C) 2009,2010 by Heinz-Bernd Eggenstein                     *
 *                                                                         *
 *   This file is part of Einstein@Home (Radio Pulsar Edition).            *
 *                                                                         *
 *   Description:                                                          *
 *   Performs harmonic summing (2nd ... 16th harmonic) of powerspectrum    *
 *   CUDA variant.                                                         *
 *                                                                         *
 *   Einstein@Home is free software: you can redistribute it and/or modify *
 *   it under the terms of the GNU General Public License as published     *
 *   by the Free Software Foundation, version 2 of the License.            *
 *                                                                         *
 *   Einstein@Home is distributed in the hope that it will be useful,      *
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the          *
 *   GNU General Public License for more details.                          *
 *                                                                         *
 *   You should have received a copy of the GNU General Public License     *
 *   along with Einstein@Home. If not, see <http://www.gnu.org/licenses/>. *
 *                                                                         *
 ***************************************************************************/

#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>

#include "demod_binary_hs_cuda.cuh"
#include "cuda_utilities.h"
#include "../../erp_utilities.h"
#include "../../hs_common.h"
#include "../../demod_binary.h"


#define HS_BLOCKSIZE 256    // must be an integer power of 2 (because of the following constraint)
#define HS_LOG_BLOCKSIZE 8  // constraint: HS_LOG_BLOCKSIZE = lrint(log2(HS_BLOCKSIZE))

#include "harmonic_summing_kernel.cuh"


float * powerspectrumHost = 0; 
int     stdmem_powerspectrum = 0;  // Flag to distinguish page-locked allocation



// module global variables

hipDeviceptr_t h_lutDev; // look up tables in global device memory
hipDeviceptr_t k_lutDev;
hipDeviceptr_t thrADev;  // threshold for 1st , 2nd, 4th, 8th, 16th harmonics on device
hipDeviceptr_t sumspecDev[5] ;  /* an array of device memory pointers (NOT an array on the device!) */

hipModule_t cuModuleHS;   // device module and kernel handles
hipFunction_t kernelHarmonicSumming;
hipFunction_t kernelHarmonicSummingGaps;
hipTexRef h_lutTexRef;
hipTexRef k_lutTexRef;
hipTexRef thrATexRef;
hipTexRef powerspectrumTexRef;


int set_up_harmonic_summing(float ** sumspec,int32_t ** dirty, unsigned int * nr_pages_ptr, unsigned int fundamental_idx_hi,unsigned int harmonic_idx_hi)
{
    hipError_t cuResult = hipSuccess;
    int i;
    unsigned int nr_pages;

    // load device modules / kernels
    char modulePath[1024] = {0};
    i = resolveFilename("dbhs.dev", modulePath, 1023);
    if(i) {
	logMessage(error, true, "Couldn't retrieve HS CUDA device module path (error: %i)!\n", i);
	return(RADPUL_CUDA_LOAD_MODULE);
    }
    cuResult = hipModuleLoad(&cuModuleHS, modulePath);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Couldn't load HS CUDA device module (error: %i)!\n", cuResult);
        return(RADPUL_CUDA_LOAD_MODULE);
    }

    cuResult = hipModuleGetFunction(&kernelHarmonicSumming, cuModuleHS, "harmonic_summing_kernel");
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Couldn't get CUDA HS kernel handle (error: %i)!\n", cuResult);
        return(RADPUL_CUDA_LOOKUP_KERNEL);
    }

    cuResult = hipModuleGetFunction(&kernelHarmonicSummingGaps, cuModuleHS, "harmonic_summing_kernel_gaps");
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Couldn't get CUDA HS kernel handle (error: %i)!\n", cuResult);
        return(RADPUL_CUDA_LOOKUP_KERNEL);
    }

    // allocate memory for the harmonic summed spectra
    // in CUDA version, this includes the 1st harmonics
    // TODO : sumspec [0] has to be treated differently once the powerspectrum is left on device
    //        initially and only copied async. later
    for( i = 1; i < 5; i++)
    {
      sumspec[i] = (float *) calloc(fundamental_idx_hi, sizeof(float));
      if(sumspec[i] == NULL)
      {
        logMessage(error, true, "Couldn't allocate %d bytes of memory for sumspec at bottom level.\n", fundamental_idx_hi*sizeof(float ));
        return(RADPUL_EMEM);
      }
    }

    sumspecDev[0]=0;
    for( i =1; i  < 5 ; i++) {
        cuResult = hipMalloc(&(sumspecDev[i]), sizeof(float) * fundamental_idx_hi);
        if(cuResult != hipSuccess) {
            logMessage(error, true, "Couldn't allocate %d bytes of CUDA HS summing memory (error: %i)!\n", sizeof(float) * fundamental_idx_hi, cuResult);
            return(RADPUL_CUDA_MEM_ALLOC_DEVICE);
        }
    }

    nr_pages=(fundamental_idx_hi >> LOG_PS_PAGE_SIZE)+1;
    *nr_pages_ptr = nr_pages;
    for(i = 0; i < 5 ; i++) {
      dirty[i] = (int32_t *) calloc(nr_pages, sizeof(int32_t));
      if(dirty[i] == NULL)
      {
        logMessage(error, true, "Couldn't allocate %d bytes of memory for sumspec page flags at bottom level.\n", fundamental_idx_hi*sizeof(float ));
        return(RADPUL_EMEM);
      }

    }


    // allocate texture memory
    cuResult = hipMalloc(&h_lutDev, sizeof(int32_t) * 16);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Couldn't allocate %d bytes of CUDA HSH texture memory (error: %i)!\n", sizeof(int32_t) * 16, cuResult);
        return(RADPUL_CUDA_MEM_ALLOC_DEVICE);
    }
    cuResult = hipMalloc(&k_lutDev, sizeof(int32_t) * 16);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Couldn't allocate %d bytes of CUDA HSK texture memory (error: %i)!\n", sizeof(int32_t) * 16, cuResult);
        return(RADPUL_CUDA_MEM_ALLOC_DEVICE);
    }
    cuResult = hipMalloc(&thrADev, sizeof(float) * 5);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Couldn't allocate %d bytes of CUDA HST texture memory (error: %i)!\n", sizeof(float) * 4, cuResult);
        return(RADPUL_CUDA_MEM_ALLOC_DEVICE);
    }

    // copy LUTs to texture memory
    cuResult = hipMemcpyHtoD(h_lutDev, h_lut, sizeof(int32_t) * 16);
    if(cuResult != hipSuccess)
    {
        logMessage(error, true, "Error during CUDA host->device HSH lookup table data transfer (error: %i)\n", cuResult);
        return(RADPUL_CUDA_MEM_COPY_HOST_DEVICE);
    }
    cuResult = hipMemcpyHtoD(k_lutDev, k_lut, sizeof(int32_t) * 16);
    if(cuResult != hipSuccess)
    {
        logMessage(error, true, "Error during CUDA host->device HSK lookup table data transfer (error: %i)\n", cuResult);
        return(RADPUL_CUDA_MEM_COPY_HOST_DEVICE);
    }

    // bind textures
    cuResult = hipModuleGetTexRef(&h_lutTexRef, cuModuleHS, "h_lutTex");
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Couldn't get CUDA HSH texture handle (error: %i)!\n", cuResult);
        return(RADPUL_CUDA_MEM_COPY_HOST_DEVICE);
    }
    cuResult = hipTexRefSetAddress(NULL, h_lutTexRef, h_lutDev, sizeof(int32_t) * 16);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Couldn't bind CUDA HSH texture (error: %i)!\n", cuResult);
        return(RADPUL_CUDA_MEM_COPY_HOST_DEVICE);
    }

    cuResult = hipModuleGetTexRef(&k_lutTexRef, cuModuleHS, "k_lutTex");
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Couldn't get CUDA HSK texture handle (error: %i)!\n", cuResult);
        return(RADPUL_CUDA_MEM_COPY_HOST_DEVICE);
    }
    cuResult = hipTexRefSetAddress(NULL, k_lutTexRef, k_lutDev, sizeof(int32_t) * 16);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Couldn't bind CUDA HSK texture (error: %i)!\n", cuResult);
        return(RADPUL_CUDA_MEM_COPY_HOST_DEVICE);
    }

    cuResult = hipModuleGetTexRef(&thrATexRef, cuModuleHS, "thrATex");
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Couldn't get CUDA HST texture handle (error: %i)!\n", cuResult);
        return(RADPUL_CUDA_MEM_COPY_HOST_DEVICE);
    }
    cuResult = hipTexRefSetAddress(NULL, thrATexRef, thrADev, sizeof(float) * 5);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Couldn't bind CUDA HST texture (error: %i)!\n", cuResult);
        return(RADPUL_CUDA_MEM_COPY_HOST_DEVICE);
    }

    cuResult = hipModuleGetTexRef(&powerspectrumTexRef, cuModuleHS, "powerspectrumTex");
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Couldn't get CUDA HSP texture handle (error: %i)!\n", cuResult);
        return(RADPUL_CUDA_MEM_COPY_HOST_DEVICE);
    }


    // allocate host memory for power spectrum
    cuResult = hipMemAllocHost((void**) &powerspectrumHost , harmonic_idx_hi * sizeof(float));
    if(cuResult != hipSuccess)
    {
        logMessage(warn, true, "Couldn't allocate %d bytes of pinned host memory for power spectrum (error: %i)! Trying fallback...\n", harmonic_idx_hi * sizeof(float), cuResult);
        powerspectrumHost = (float *) calloc(harmonic_idx_hi, sizeof(float));
        if(powerspectrumHost == NULL)
        {
            logMessage(error, true, "Couldn't allocate %d bytes of memory for power spectrum!\n", harmonic_idx_hi * sizeof(float));
            return(RADPUL_CUDA_MEM_ALLOC_HOST);
        }
        // set flag to indicate conventional memory allocation
        stdmem_powerspectrum = 1;
    }
    logMessage(debug, true, "Allocated host memory for power spectrum: %i bytes\n", sizeof(float) * harmonic_idx_hi);


    return 0;
}



int tear_down_harmonic_summing(float ** sumspec, int32_t ** dirty)
{
    hipError_t cuResult = hipSuccess;
    int i;

    // clean up. (0th element is powerspectrum, freed separately)
    for(i = 1; i < 5; i++) {
      free(sumspec[i]);
    }

    for(i = 0; i < 5; i++) {
      free(dirty[i]);
    }

    for( i =1; i  < 5 ; i++) {
        cuResult = hipFree(sumspecDev[i]);
        if(cuResult != hipSuccess) {
            logMessage(error, true, "Error freeing CUDA HS device memory (error: %d)\n", cuResult);
            return(RADPUL_CUDA_MEM_FREE_DEVICE);
        }
    }

    // unbind textures (is this the only/recommended way?!)
    cuResult = hipTexRefSetAddress(NULL, h_lutTexRef, NULL, 0);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Couldn't unbind CUDA HSH texture (error: %i)!\n", cuResult);
        return(RADPUL_CUDA_MEM_FREE_DEVICE);
    }
    cuResult = hipTexRefSetAddress(NULL, k_lutTexRef, NULL, 0);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Couldn't unbind CUDA HSK texture (error: %i)!\n", cuResult);
        return(RADPUL_CUDA_MEM_FREE_DEVICE);
    }
    cuResult = hipTexRefSetAddress(NULL, thrATexRef, NULL, 0);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Couldn't unbind CUDA HST texture (error: %i)!\n", cuResult);
        return(RADPUL_CUDA_MEM_FREE_DEVICE);
    }

    // free texture memory
    cuResult = hipFree(h_lutDev);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error freeing CUDA HSH texture memory (error: %d)\n", cuResult);
        return(RADPUL_CUDA_MEM_FREE_DEVICE);
    }
    cuResult = hipFree(k_lutDev);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error freeing CUDA HSK texture memory (error: %d)\n", cuResult);
        return(RADPUL_CUDA_MEM_FREE_DEVICE);
    }
    cuResult = hipFree(thrADev);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error freeing CUDA HST texture memory (error: %d)\n", cuResult);
        return(RADPUL_CUDA_MEM_FREE_DEVICE);
    }


    if(stdmem_powerspectrum) {
        free(powerspectrumHost);
    }
    else {
        cuResult = hipHostFree(powerspectrumHost);
        if(cuResult != hipSuccess) {
            logMessage(error, true, "Error deallocating CUDA pinned host powerspectrum memory (error: %i)\n", cuResult);
            return(RADPUL_CUDA_MEM_FREE_HOST);
        }
    }


    return 0;
}



int run_harmonic_summing(float ** sumspec, int32_t ** dirty, unsigned int nr_pages, DIfloatPtr  powerspectrum_dip , unsigned int 
window_2,unsigned int fundamental_idx_hi,unsigned int harmonic_idx_hi, float *thresholds)
{
      unsigned int l1,l2,i,j,k;

      hipError_t cuResult = hipSuccess;
      dim3 dg1,dg2;
      dim3 db1,db2;
      int nr_pages_total = nr_pages * 5;
   
      float * powerspectrum = powerspectrumHost; // global variable

      /* borders for main kernel computation in a 16 index grid */
      /* for simplicity we always start at the left border of the spectrum,
        the kernel itself will take care of the window_2 offset */
      l1= 0;
      /* the number of main kernel blocks of width 16 that is needed to fully cover
         the spectrum up to index harmonic_idx_hi -1 (inclusive) */
      l2= ((harmonic_idx_hi -1 + 8) >> 4) +1 ;

      hipDeviceptr_t powerspectrumDev=powerspectrum_dip.device_ptr;;
      hipDeviceptr_t dirtyDev;
      hipStream_t stream[2];
      int32_t * dirtyTmp; 
      hipStreamCreateWithFlags(&stream[0], 0);
      hipStreamCreateWithFlags(&stream[1], 0);


      // add powerspectrum as first spectra element

      // TODO : sumspec [0] has to be treated differently once the powerspectrum is left on device
      //        initially and only copied (possibly async.) later

      sumspec[0] = powerspectrum;


      /* allocate sumspec arrays on device */
      sumspecDev[0] = powerspectrumDev;  
      for( i =1; i  < 5 ; i++) {
          cuResult = hipMemsetD8(sumspecDev[i], 0, sizeof(float) * fundamental_idx_hi);
          if(cuResult != hipSuccess) {
              logMessage(error, true, "Couldn't erase %d bytes of CUDA HS summing memory (error: %i)!\n", sizeof(float) * fundamental_idx_hi, cuResult);
              return(RADPUL_CUDA_MEM_COPY_HOST_DEVICE);
          }
      }

      cuResult = hipMalloc(&dirtyDev, sizeof(int32_t) *  nr_pages_total);
      if(cuResult != hipSuccess) {
          logMessage(error, true, "Couldn't allocate %d bytes of CUDA HS summing memory (error: %i)!\n", sizeof(int32_t) * 
nr_pages_total,cuResult);
              return(RADPUL_CUDA_MEM_ALLOC_DEVICE);
      }
      cuResult = hipMemsetD8(dirtyDev, 0, sizeof(int32_t) * nr_pages_total);
      if(cuResult != hipSuccess) {
              logMessage(error, true, "Couldn't erase %d bytes of CUDA HS summing memory (error: %i)!\n", sizeof(int32_t) * nr_pages_total, cuResult);
              return(RADPUL_CUDA_MEM_COPY_HOST_DEVICE);
      }
 


      /* copy thresholds to device */


      cuResult = hipMemcpyHtoD(thrADev, thresholds, sizeof(float) * 5);
      if(cuResult != hipSuccess)
      {
          logMessage(error, true, "Error during CUDA host->device HS thresholds data transfer (error: %i)\n", cuResult);
          return(RADPUL_CUDA_MEM_COPY_HOST_DEVICE);
      }

      /* the powerspectrum is a not so obvious candidate for texture memory. Each kernel
         reads powerspectrum[i*k /16 +0.5] for k = 1..15, so many cells are read more than
         once, so caching should be beneficial. Cached access should also benefit from data locality */
      cuResult = hipTexRefSetAddress(NULL, powerspectrumTexRef, powerspectrumDev, sizeof(float) * harmonic_idx_hi);
      if(cuResult != hipSuccess) {
          logMessage(error, true, "Couldn't bind CUDA HSP texture (error: %i)!\n", cuResult);
          return(RADPUL_CUDA_MEM_COPY_HOST_DEVICE);
      }

/* Execute kernel to perform harmonic summing (with some gaps where sumspec target values would overlap */

/* somehow this seems to work better than y=16 , x = (l2-l1)/16 */
/* anyway we have to use a 2 dim because index in each dim is limited */
      dg1.y=(l2-l1)/HS_BLOCKSIZE ;

      /* add one block if not perfectly aligned */
      if((l2-l1) % HS_BLOCKSIZE !=0 ) {
        dg1.y++;
      }
      dg1.x=16; // rather arbitrarily, there is no algorithmic reason for the value 16
      dg1.z=1;

      db1.x=HS_BLOCKSIZE;
      db1.y=1;
      db1.z=1;

      logMessage(debug, true, "Executing harmonic summing CUDA kernel (%u threads each in %u blocks)...\n", db1.x, dg1.x * dg1.y * dg1.z);

      /* the lowest index i for which sumspec[h][i/(1<<h)+0.5] is computed by the kernels*/

      /* first kernel in first stream*/
      /* TODO: check effect on performance of having concurrent streams */

      // prepare parameters
      int kernelParamOffset = 0;
      for(i = 1; i < 5; ++i) {
          KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(sumspecDev[i]));
          cuResult = cuParamSetv(kernelHarmonicSumming, kernelParamOffset, &(sumspecDev[i]), sizeof(sumspecDev[i]));
          kernelParamOffset += sizeof(sumspecDev[i]);
          if(cuResult != hipSuccess) {
              logMessage(error, true, "Error during CUDA HS kernel parameter setup (error: %d)\n", cuResult);
              return(RADPUL_CUDA_KERNEL_PREPARE);
          }
      }

      KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(dirtyDev));
      cuResult = cuParamSetv(kernelHarmonicSumming, kernelParamOffset, &dirtyDev, sizeof(dirtyDev));
      kernelParamOffset += sizeof(dirtyDev);
      if(cuResult != hipSuccess) {
          logMessage(error, true, "Error during CUDA HS kernel parameter setup (error: %d)\n", cuResult);
          return(RADPUL_CUDA_KERNEL_PREPARE);
      }

      KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(powerspectrumDev));
      cuResult = cuParamSetv(kernelHarmonicSumming, kernelParamOffset, &powerspectrumDev, sizeof(powerspectrumDev));
      kernelParamOffset += sizeof(powerspectrumDev);
      if(cuResult != hipSuccess) {
          logMessage(error, true, "Error during CUDA HS kernel parameter setup (error: %d)\n", cuResult);
          return(RADPUL_CUDA_KERNEL_PREPARE);
      }

      KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(window_2));
      cuResult = cuParamSeti(kernelHarmonicSumming, kernelParamOffset, window_2);
      kernelParamOffset += sizeof(window_2);
      if(cuResult != hipSuccess) {
          logMessage(error, true, "Error during CUDA HS kernel parameter setup (error: %d)\n", cuResult);
          return(RADPUL_CUDA_KERNEL_PREPARE);
      }

      KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(fundamental_idx_hi));
      cuResult = cuParamSeti(kernelHarmonicSumming, kernelParamOffset, fundamental_idx_hi);
      kernelParamOffset += sizeof(fundamental_idx_hi);
      if(cuResult != hipSuccess) {
          logMessage(error, true, "Error during CUDA HS kernel parameter setup (error: %d)\n", cuResult);
          return(RADPUL_CUDA_KERNEL_PREPARE);
      }

      KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(harmonic_idx_hi));
      cuResult = cuParamSeti(kernelHarmonicSumming, kernelParamOffset, harmonic_idx_hi);
      kernelParamOffset += sizeof(harmonic_idx_hi);
      if(cuResult != hipSuccess) {
          logMessage(error, true, "Error during CUDA HS kernel parameter setup (error: %d)\n", cuResult);
          return(RADPUL_CUDA_KERNEL_PREPARE);
      }

      cuResult = cuParamSetSize(kernelHarmonicSumming, kernelParamOffset);
      if(cuResult != hipSuccess) {
          logMessage(error, true, "Error finalizing CUDA HS kernel parameter setup (error: %d)\n", cuResult);
          return(RADPUL_CUDA_KERNEL_PREPARE);
      }

      // prepare block size
      cuResult = cuFuncSetBlockShape(kernelHarmonicSumming, db1.x, db1.y, db1.z);
      if(cuResult != hipSuccess) {
          logMessage(error, true, "Error during CUDA HS kernel block setup (error: %d)\n", cuResult);
          return(RADPUL_CUDA_KERNEL_PREPARE);
      }

      // launch kernel grid
      cuResult = cuLaunchGridAsync(kernelHarmonicSumming, dg1.x, dg1.y, stream[0]);
      if(cuResult != hipSuccess) {
          logMessage(error, true, "Error launching CUDA HS kernel (error: %d)\n", cuResult);
          return(RADPUL_CUDA_KERNEL_INVOKE);
      }

/* execute second kernel, this time to fill the gaps */

      l1=0;
      /* the number of gap kernel blocks (where each block covers 2 segments of length 8 indices)
      l2= (((harmonic_idx_hi -1 + 12) >> 4 ) +1) /*>> 1*/ ; /* TODO CHECK!!!!!*/

      dg2.y=(l2-l1)/HS_BLOCKSIZE;
      /* add one if not perfectly aligned to HS_BLOCKSIZE */
      if((l2-l1) % HS_BLOCKSIZE != 0) {
        dg2.y++;
      }
      dg2.x=16;  /* again, 16 is rather arbitrary */
      dg2.z=1;

      db2.x=HS_BLOCKSIZE/2;  /* sic! it is essential that the blocksize is half that of the first kernel */
      db2.y=1;
      db2.z=1;

      logMessage(debug, true, "Executing harmonic summing gaps CUDA kernel (%u threads each in %u blocks)...\n", db2.x, dg2.x * dg2.y * dg2.z);

      // prepare parameters
      kernelParamOffset = 0;
      for(i = 1; i < 5; ++i) {
          KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(sumspecDev[i]));
          cuResult = cuParamSetv(kernelHarmonicSummingGaps, kernelParamOffset, &sumspecDev[i], sizeof(sumspecDev[i]));
          kernelParamOffset += sizeof(sumspecDev[i]);
          if(cuResult != hipSuccess) {
              logMessage(error, true, "Error during CUDA HSG kernel parameter setup (error: %d)\n", cuResult);
              return(RADPUL_CUDA_KERNEL_PREPARE);
          }
      }
      
      KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(dirtyDev));
      cuResult = cuParamSetv(kernelHarmonicSummingGaps, kernelParamOffset, &dirtyDev, sizeof(dirtyDev));
      kernelParamOffset += sizeof(dirtyDev);
      if(cuResult != hipSuccess) {
          logMessage(error, true, "Error during CUDA HS kernel parameter setup (error: %d)\n", cuResult);
          return(RADPUL_CUDA_KERNEL_PREPARE);
      }
      
      KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(powerspectrumDev));
      cuResult = cuParamSetv(kernelHarmonicSummingGaps, kernelParamOffset, &powerspectrumDev, sizeof(powerspectrumDev));
      kernelParamOffset += sizeof(powerspectrumDev);
      if(cuResult != hipSuccess) {
          logMessage(error, true, "Error during CUDA HSG kernel parameter setup (error: %d)\n", cuResult);
          return(RADPUL_CUDA_KERNEL_PREPARE);
      }

      KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(window_2));
      cuResult = cuParamSeti(kernelHarmonicSummingGaps, kernelParamOffset, window_2);
      kernelParamOffset += sizeof(window_2);
      if(cuResult != hipSuccess) {
          logMessage(error, true, "Error during CUDA HSG kernel parameter setup (error: %d)\n", cuResult);
          return(RADPUL_CUDA_KERNEL_PREPARE);
      }

      KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(fundamental_idx_hi));
      cuResult = cuParamSeti(kernelHarmonicSummingGaps, kernelParamOffset, fundamental_idx_hi);
      kernelParamOffset += sizeof(fundamental_idx_hi);
      if(cuResult != hipSuccess) {
          logMessage(error, true, "Error during CUDA HSG kernel parameter setup (error: %d)\n", cuResult);
          return(RADPUL_CUDA_KERNEL_PREPARE);
      }

      KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(harmonic_idx_hi));
      cuResult = cuParamSeti(kernelHarmonicSummingGaps, kernelParamOffset, harmonic_idx_hi);
      kernelParamOffset += sizeof(harmonic_idx_hi);
      if(cuResult != hipSuccess) {
          logMessage(error, true, "Error during CUDA HSG kernel parameter setup (error: %d)\n", cuResult);
          return(RADPUL_CUDA_KERNEL_PREPARE);
      }

      cuResult = cuParamSetSize(kernelHarmonicSummingGaps, kernelParamOffset);
      if(cuResult != hipSuccess) {
          logMessage(error, true, "Error finalizing CUDA HSG kernel parameter setup (error: %d)\n", cuResult);
          return(RADPUL_CUDA_KERNEL_PREPARE);
      }

      // prepare block size
      cuResult = cuFuncSetBlockShape(kernelHarmonicSummingGaps, db2.x, db2.y, db2.z);
      if(cuResult != hipSuccess) {
          logMessage(error, true, "Error during CUDA HSG kernel block setup (error: %d)\n", cuResult);
          return(RADPUL_CUDA_KERNEL_PREPARE);
      }

      // launch kernel grid
      cuResult = cuLaunchGridAsync(kernelHarmonicSummingGaps, dg2.x, dg2.y, stream[1]);
      if(cuResult != hipSuccess) {
          logMessage(error, true, "Error launching CUDA HSG kernel (error: %d)\n", cuResult);
          return(RADPUL_CUDA_KERNEL_INVOKE);
      }

      // wait until all device processing finished (errors should indicate earlier launch failures)
      cuResult = hipCtxSynchronize();
      if(cuResult != hipSuccess) {
          logMessage(error, true, "Error during CUDA HS/HSG kernel launch and/or device synchronization (error: %d)\n", cuResult);
          return(RADPUL_CUDA_KERNEL_INVOKE);
      }

      // destroy both streams
      hipStreamDestroy(stream[0]);
      hipStreamDestroy(stream[1]);
      // unbind texture
      cuResult = hipTexRefSetAddress(NULL, powerspectrumTexRef, NULL, 0);
      if(cuResult != hipSuccess) {
          logMessage(error, true, "Couldn't unbind CUDA HSP texture (error: %i)!\n", cuResult);
          return(RADPUL_CUDA_MEM_FREE_DEVICE);
      }


      // copy back dirty page flags to memory

      dirtyTmp = (int32_t*) malloc (nr_pages_total * sizeof(int32_t));
      if(dirtyTmp == NULL) { 
          logMessage(error, true, "Couldn't allocate %d bytes of memory for temp mem (HS).\n", nr_pages_total * sizeof(int32_t) );
          return(RADPUL_EMEM);
      };
      cuResult = hipMemcpyDtoH(dirtyTmp,dirtyDev,sizeof(int32_t) * nr_pages_total);
      if(cuResult != hipSuccess) {  
          logMessage(error, true, "Error during CUDA device->host HS data transfer (dirty) (error: %d)\n", cuResult);
          return(RADPUL_CUDA_MEM_COPY_DEVICE_HOST);
      }


      int dirty_idx_min[5] = {0,0,0,0,0};
      int dirty_idx_max[5] = {0,0,0,0,0};
      int d,d_min,d_max;
      

      k=0;
      for(i=0 ; i < 5 ; i++) {
        d_min=nr_pages;
        d_max=-1;

        // find the first dirty page
	for(j=0; j < nr_pages ; j++) {
	   d=dirty[i][j] = dirtyTmp[k++];
	   if(d!=0) {
		d_min = j ; 
               d_max = j;
               j++; 
		break;
	   }
        }
        // go thru the rest and record the last dirty page we find

        for(    ; j < nr_pages ; j++) {
           d=dirty[i][j] = dirtyTmp[k++];
           if(d!=0) {
                d_max = j ;                          
           }
        }

        dirty_idx_min[i]=d_min;
        dirty_idx_max[i]=d_max;

      } 		

      free(dirtyTmp);
      cuResult = hipFree(dirtyDev);
      if(cuResult != hipSuccess) {
          logMessage(error, true, "Error freeing CUDA HS device memory (error: %d)\n", cuResult);
          return(RADPUL_CUDA_MEM_FREE_DEVICE);
      }



      /* copy back the results from the CUDA kernel.
       * make sure to copy only those cells from sumspec
       * (including the "1st harmonics" powerspectrum itself)
       * that have a chance to include a candidate that makes it to the toplist
       *
       * TODO : look at possibility to copy only a subarray
       */

      for( i =0; i  < 5 ; i++) {
          /* no need to copy anything if there is no potential candidate at all */
          if (dirty_idx_max[i]!=-1) {
                  size_t seg_offset =  dirty_idx_min[i] << LOG_PS_PAGE_SIZE;
                  size_t seg_length = (dirty_idx_max[i]-dirty_idx_min[i] +1)  << LOG_PS_PAGE_SIZE;		
                  // clip the segment to be copied at the max length of the array
                  size_t seg_length_limit = fundamental_idx_hi - seg_offset;
                  if (seg_length > seg_length_limit) {
                      seg_length = seg_length_limit;
                  }

		  // do some pointer arithmetic to get the the right subsegment of memory to copy
		  cuResult = hipMemcpyDtoH(sumspec[i]+seg_offset , (hipDeviceptr_t) (((float*) sumspecDev[i])+seg_offset)   , sizeof(float) * seg_length);
		  if(cuResult != hipSuccess) {
		      logMessage(error, true, "Error during CUDA device->host HS data transfer (error: %d)\n", cuResult);
		      return(RADPUL_CUDA_MEM_COPY_DEVICE_HOST);
                  }
          }
      }





  return 0;
}
