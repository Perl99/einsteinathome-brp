#include "hip/hip_runtime.h"
/***************************************************************************
 *   Copyright (C) 2010 by Oliver Bock                                     *
 *   oliver.bock[AT]aei.mpg.de                                             *
 *                                                                         *
 *   This file is part of Einstein@Home (Radio Pulsar Edition).            *
 *                                                                         *
 *   Description:                                                          *
 *   Demodulates dedispersed time series using a bank of orbital           *
 *   parameters. After this step, an FFT of the resampled time series is   *
 *   searched for pulsed, periodic signals by harmonic summing.            *
 *                                                                         *
 *   Einstein@Home is free software: you can redistribute it and/or modify *
 *   it under the terms of the GNU General Public License as published     *
 *   by the Free Software Foundation, version 2 of the License.            *
 *                                                                         *
 *   Einstein@Home is distributed in the hope that it will be useful,      *
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the          *
 *   GNU General Public License for more details.                          *
 *                                                                         *
 *   You should have received a copy of the GNU General Public License     *
 *   along with Einstein@Home. If not, see <http://www.gnu.org/licenses/>. *
 *                                                                         *
 ***************************************************************************/

#include "demod_binary_cuda.h"

#include <stdlib.h>
#include <math.h>
#include <gsl/gsl_math.h>
#include "../../demod_binary.h"
#include "../../erp_utilities.h"
#include <hip/hip_runtime.h>
#include "cuda_utilities.h"

// TODO: do we wanna keep those global (or use proper C++, or pass them around)?
hipDevice_t cuDevice = NULL;                 // CUDA device pointer
hipCtx_t cuContext = NULL;               // CUDA context we work in
hipModule_t cuModuleMain;                    // Device module, kernel and symbol handles
hipFunction_t kernelTimeSeriesModulation;
hipFunction_t kernelTimeSeriesLengthModulated;
hipFunction_t kernelTimeSeriesResampling;
hipFunction_t kernelTimeSeriesMeanReduction;
hipFunction_t kernelTimeSeriesPadding;
hipFunction_t kernelPowerSpectrum;
hipDeviceptr_t timeSeriesLengthDeviceBuffer;
hipError_t cuResult = hipSuccess;         // CUDA return results (driver API)
hipfftResult_t cufResult = HIPFFT_SUCCESS;  // CUFFT return results
int cudDriverVersion = 0;                 // Version of the installed CUDA driver (not runtime)
unsigned int cudApiVersion = 0;           // Version of the installed CUDA driver API
hipDeviceProp_t cudDeviceProperties;       // CUDA device properties

hipDeviceptr_t originalTimeSeriesDeviceBuffer;    // Original time series device buffer
hipDeviceptr_t modTimeOffsetsDeviceBuffer;        // Modulated time offsets device buffer
hipDeviceptr_t timeSeriesMeanDeviceBuffer;        // Sum-reduction buffer for time series mean value
hipDeviceptr_t sinLUTDeviceBuffer;                // sin lookup table device buffer
hipDeviceptr_t cosLUTDeviceBuffer;                // cos lookup table device buffer

hipfftHandle cufPlan;                      // FFT plan handle

#include "demod_binary_cuda.cuh"

// macro for fft padding (based on powerspectrum kernel's blocksize, defined in demod_binary_cuda.cuh)
#define PADDED_FFT_SIZE(fftsize) CUDA_FFT_BLOCKDIM_X * (unsigned int) ceil((float)fftsize / (float)CUDA_FFT_BLOCKDIM_X)


int initialize_cuda(int cudDeviceIdGiven, int *cudDeviceIdPtr)
{
    int i;
    int res;
    // has the CUDA device been set already (doing this a second time results in an error)?
    static bool cudaDeviceNotSet = true;
    static int cudDeviceId = *cudDeviceIdPtr;

    if (cudaDeviceNotSet) {
        // initialize driver API
        cuResult = hipInit(0);
        if(cuResult != hipSuccess) {
            logMessage(error, true, "Couldn't initialize CUDA driver API (error: %i)!\n", cuResult);
            return(RADPUL_CUDA_DRIVER_INIT);
        }

#ifdef BOINCIFIED
        // if controlled by a BOINC client, check device_num in init_data.xml
        // terminate if we couldn't find one there.
        //
        if (!cudDeviceIdGiven && !running_standalone()) {

            // we already checked the command line for the device number, so pass empty command line here
            res = boinc_get_cuda_device_id(0,NULL, & cudDeviceId);
            if (res) {
                logMessage(error, true, "No suitable CUDA device available!\n");
                return(RADPUL_CUDA_DEVICE_FIND);
            } else {
                cudDeviceIdGiven = 1;
            }
        }
#endif

        // if no device was explicitly specified so far, find best suitable CUDA device
        if (!cudDeviceIdGiven) {

            // find appropriate CUDA device ourself;
            logMessage(debug, true, "No (valid) device ID passed via command line. Determining suitable device... \n");
            cudDeviceId = findBestFreeDevice(0,0,0,0);
            if(cudDeviceId < 0) {
                logMessage(error, true, "No suitable CUDA device available!\n");
                return(RADPUL_CUDA_DEVICE_FIND);
            }
        }

        // update caller's device ID value
        *cudDeviceIdPtr = cudDeviceId;

        // Get handle for requested device
        cuResult = hipDeviceGet(&cuDevice, cudDeviceId);
        if(cuResult != hipSuccess) {
            logMessage(error, true, "Couldn't acquire CUDA device #%i (error: %i)!\n", cudDeviceId, cuResult);
            return(RADPUL_CUDA_DEVICE_SET);
        }

        cudaDeviceNotSet = false;
    }

    // acquire device (set thread scheduling to yield/block during GPU execution: increases latency but reduces CPU usage -> BOINC!)
    cuResult = hipCtxCreate(&cuContext, hipDeviceScheduleBlockingSync, cudDeviceId);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Failed to enable CUDA thread yielding for device #%i (error: %i)! Sorry, will try to occupy one CPU core...\n",
                   cudDeviceId, cuResult);

        // retry with auto scheduling
        cuResult = hipCtxCreate(&cuContext, hipDeviceScheduleAuto, cudDeviceId);
        if(cuResult != hipSuccess) {
            logMessage(error, true, "Couldn't acquire CUDA context of device #%i (error: %i)!\n", cudDeviceId, cuResult);
            return(RADPUL_CUDA_DEVICE_SET);
        }
    }

    // show some details if possible
    int compcapMajor = 0;
    int compcapMinor = 0;
    int multiProcessorCount = 0;
    int coreCount = 0;
    int clockRate = 0;
    int flopsPerClockTick = 0;
    char deviceName[256] = {0};

    logMessage(info, true, "CUDA global memory status (initial GPU state, including context):\n");
    printDeviceGlobalMemStatus(info, true);

    // number of multi processors
    cuResult = hipDeviceGetAttribute(&multiProcessorCount, hipDeviceAttributeMultiprocessorCount, cuDevice);
    if(cuResult != hipSuccess) {
        logMessage(warn, true, "Couldn't retrieve multiprocessor count property of device #%i (error: %i)! Trying next one...\n", cudDeviceId, cuResult);
    }

    // clock rate
    cuResult = hipDeviceGetAttribute(&clockRate, hipDeviceAttributeClockRate, cuDevice);
    if(cuResult != hipSuccess) {
        logMessage(warn, true, "Couldn't retrieve clock rate property of device #%i (error: %i)! Trying next one...\n", cudDeviceId, cuResult);
    }

    // compute capability
    cuResult = hipDeviceComputeCapability (&compcapMajor, &compcapMinor, cuDevice);
    if(cuResult != hipSuccess) {
        logMessage(warn, true, "Couldn't retrieve compute capability of device #%i (error: %i)! Trying next one...\n", cudDeviceId, cuResult);
    }

    // assign proper number of cores
    if(compcapMajor == 1) {
        coreCount = multiProcessorCount * 8;
        flopsPerClockTick = 3;
    }
    else if(compcapMajor == 2 && compcapMinor == 0) {
        coreCount = multiProcessorCount * 32;
        flopsPerClockTick = 2;
    }
    else if(compcapMajor == 2 && compcapMinor >= 0) {
        coreCount = multiProcessorCount * 48;
        flopsPerClockTick = 2;
    }
    else if(compcapMajor == 3) {
        coreCount = multiProcessorCount * 192;
        flopsPerClockTick = 2;
    }
    else if(compcapMajor == 5) {
        coreCount = multiProcessorCount * 128;
        flopsPerClockTick = 2;
    }


    // name
    cuResult = hipDeviceGetName(deviceName, 256, cuDevice);
    if(cuResult != hipSuccess) {
        logMessage(debug, true, "Couldn't retrieve name of device #%i (error: %i)!\n", cudDeviceId, cuResult);
        strcpy(deviceName, "UNKNOWN");
    }

    // check if our device is a "real" device
    if(compcapMajor == 9999 || compcapMinor == 9999) {
        logMessage(error, true, "Error acquiring \"real\" CUDA device!\n");
        logMessage(error, false, "The acquired device is a \"%s\"\n", deviceName);
        return(RADPUL_CUDA_EMULATION_MODE);
    }
    else {
        logMessage(info, true, "Using CUDA device #%i \"%s\" (%i CUDA cores / %.2f GFLOPS)\n",
                   cudDeviceId, deviceName, coreCount, coreCount * clockRate * flopsPerClockTick * 1e-6);
    }

    // determine CUDA driver version
    cuResult = hipDriverGetVersion(&cudDriverVersion);
    if(cuResult != hipSuccess) {
        logMessage(warn, true, "Couldn't retrieve CUDA driver version (error: %i)!\n", cuResult);
    }
    else {
        logMessage(info, true, "Version of installed CUDA driver: %i\n", cudDriverVersion);
    }

    // determine CUDA driver API version
    cuResult = hipCtxGetApiVersion(NULL, &cudApiVersion);
    if(cuResult != hipSuccess) {
        logMessage(warn, true, "Couldn't retrieve CUDA driver API version (error: %i)!\n", cuResult);
    }
    else {
        logMessage(info, true, "Version of CUDA driver API used: %u\n", cudApiVersion);
    }

    // load device modules / kernels
    char modulePath[1024] = {0};
    i = resolveFilename("db.dev", modulePath, 1023);
    if(i) {
        logMessage(error, true, "Couldn't retrieve main CUDA device module path (error: %i)!\n", i);
        return(RADPUL_CUDA_LOAD_MODULE);
    }
    cuResult = hipModuleLoad(&cuModuleMain, modulePath);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Couldn't load main CUDA device module (error: %i)!\n", cuResult);
        return(RADPUL_CUDA_LOAD_MODULE);
    }

    cuResult = hipModuleGetFunction(&kernelTimeSeriesModulation, cuModuleMain, "time_series_modulation");
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Couldn't get CUDA TSM kernel handle (error: %i)!\n", cuResult);
        return(RADPUL_CUDA_LOOKUP_KERNEL);
    }

    cuResult = hipModuleGetFunction(&kernelTimeSeriesLengthModulated, cuModuleMain, "time_series_length_modulated");
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Couldn't get CUDA TSLM kernel handle (error: %i)!\n", cuResult);
        return(RADPUL_CUDA_LOOKUP_KERNEL);
    }

    cuResult = hipModuleGetFunction(&kernelTimeSeriesResampling, cuModuleMain, "time_series_resampling");
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Couldn't get CUDA TSR kernel handle (error: %i)!\n", cuResult);
        return(RADPUL_CUDA_LOOKUP_KERNEL);
    }

    cuResult = hipModuleGetFunction(&kernelTimeSeriesMeanReduction, cuModuleMain, "time_series_mean_reduction");
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Couldn't get CUDA TSMR kernel handle (error: %i)!\n", cuResult);
        return(RADPUL_CUDA_LOOKUP_KERNEL);
    }

    cuResult = hipModuleGetFunction(&kernelTimeSeriesPadding, cuModuleMain, "time_series_padding");
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Couldn't get CUDA TSP kernel handle (error: %i)!\n", cuResult);
        return(RADPUL_CUDA_LOOKUP_KERNEL);
    }

    cuResult = hipModuleGetFunction(&kernelPowerSpectrum, cuModuleMain, "fft_powerspectrum");
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Couldn't get CUDA PS kernel handle (error: %i)!\n", cuResult);
        return(RADPUL_CUDA_LOOKUP_KERNEL);
    }

    return 0;
}


int set_up_resampling(DIfloatPtr input_dip, DIfloatPtr *output_dip, const RESAMP_PARAMS *const params, float *sinLUTsamples, float *cosLUTsamples)
{
    float * input = input_dip.host_ptr; // original time series is on host

    hipDeviceptr_t resampledTimeSeriesDeviceBuffer;   // Resampled time series device buffer (also used for FFT in-/output)

    // sanity check
    if(params->nsamples_unpadded % CUDA_RESAMP_REDUCTION_BLOCKDIM_X != 0) {
        logMessage(error, true, "The time series length %i isn't an integer multiple of the CUDA block size %i!\n", params->nsamples_unpadded, CUDA_RESAMP_REDUCTION_BLOCKDIM_X);
        return(RADPUL_EVAL);
    }

    // allocate device memory for original time series
    cuResult = hipMalloc(&originalTimeSeriesDeviceBuffer, sizeof(float) * params->nsamples_unpadded);
    if(cuResult != hipSuccess)
    {
        logMessage(error, true, "Error allocating original time series device memory: %i bytes (error: %i)\n", sizeof(float) * params->nsamples_unpadded, cuResult);
        return(RADPUL_CUDA_MEM_ALLOC_DEVICE);
    }
    logMessage(debug, true, "Allocated original time series device memory: %i bytes\n", sizeof(float) * params->nsamples_unpadded);

    // allocate device memory for modulated time offsets
    cuResult = hipMalloc(&modTimeOffsetsDeviceBuffer, sizeof(float) * params->nsamples_unpadded);
    if(cuResult != hipSuccess)
    {
        logMessage(error, true, "Error allocating modulated time offsets device memory: %i bytes (error: %i)\n", sizeof(float) * params->nsamples_unpadded, cuResult);
        return(RADPUL_CUDA_MEM_ALLOC_DEVICE);
    }
    logMessage(debug, true, "Allocated modulated time offsets device memory: %i bytes\n", sizeof(float) * params->nsamples_unpadded);

    // increase FFT buffer length such that it matches the powerspectrum kernel's blocklength (no further control flow required in kernel)
    const unsigned int fft_size_padded = PADDED_FFT_SIZE(params->fft_size);

    // allocate device memory for resampled time series (we use hipfftComplex*fft_size_padded here as we reuse the buffer later on for the FFT)
    cuResult = hipMalloc(&resampledTimeSeriesDeviceBuffer, sizeof(hipfftComplex) * fft_size_padded);
    if(cuResult != hipSuccess)
    {
        logMessage(error, true, "Error allocating modulated time series device memory: %i bytes (error: %d)\n", sizeof(hipfftComplex) * fft_size_padded, cuResult);
        return(RADPUL_CUDA_MEM_ALLOC_DEVICE);
    }
    logMessage(debug, true, "Allocated modulated time series device memory: %i bytes\n", sizeof(hipfftComplex) * fft_size_padded);

    // allocate device memory for time series mean sum reduction (we need two separate buffers, each half used alternately)
    cuResult = hipMalloc(&timeSeriesMeanDeviceBuffer, sizeof(float) * (params->nsamples_unpadded/CUDA_RESAMP_REDUCTION_BLOCKDIM_X*2));
    if(cuResult != hipSuccess)
    {
        logMessage(error, true, "Error allocating modulated time series mean reduction device memory: %i bytes (error: %i)\n", sizeof(float) * (params->nsamples_unpadded/CUDA_RESAMP_REDUCTION_BLOCKDIM_X*2), cuResult);
        return(RADPUL_CUDA_MEM_ALLOC_DEVICE);
    }
    logMessage(debug, true, "Allocated time series mean reduction device memory: %i bytes\n", sizeof(float) * (params->nsamples_unpadded/CUDA_RESAMP_REDUCTION_BLOCKDIM_X*2));

    // transfer original time series data to device
    cuResult = hipMemcpyHtoD(originalTimeSeriesDeviceBuffer, input, sizeof(float) * params->nsamples_unpadded);
    if(cuResult != hipSuccess)
    {
        logMessage(error, true, "Error during CUDA host->device original time series data transfer (error: %i)\n", cuResult);
        return(RADPUL_CUDA_MEM_COPY_HOST_DEVICE);
    }
    logMessage(debug, true, "CUDA host->device original time series data transfer successful...\n");

    // lookup constant memory symbols
    hipDeviceptr_t cudSymbol;

    // transfer sin lookup table data to device
    cuResult = hipModuleGetGlobal(&cudSymbol, NULL, cuModuleMain, "constSinSamples");
    if(cuResult != hipSuccess) {
            logMessage(error, true, "Couldn't get CUDA CSS symbol handle (error: %i)!\n", cuResult);
            return(RADPUL_CUDA_LOOKUP_SYMBOL);
    }
    cuResult = hipMemcpyHtoD(cudSymbol, sinLUTsamples, ERP_SINCOS_LUT_SIZE * sizeof(float));
    if(cuResult != hipSuccess)
    {
        logMessage(error, true, "Error during CUDA host->device sin lookup table data transfer (error: %i)\n", cuResult);
        return(RADPUL_CUDA_MEM_COPY_HOST_DEVICE);
    }
    logMessage(debug, true, "CUDA host->device sin lookup table data transfer successful...\n");

    // transfer cos lookup table data to device
    cuResult = hipModuleGetGlobal(&cudSymbol, NULL, cuModuleMain, "constCosSamples");
    if(cuResult != hipSuccess) {
            logMessage(error, true, "Couldn't get CUDA CCS symbol handle (error: %i)!\n", cuResult);
            return(RADPUL_CUDA_LOOKUP_SYMBOL);
    }
    cuResult = hipMemcpyHtoD(cudSymbol, cosLUTsamples, ERP_SINCOS_LUT_SIZE * sizeof(float));
    if(cuResult != hipSuccess)
    {
        logMessage(error, true, "Error during CUDA host->device cos lookup table data transfer (error: %d)\n", cuResult);
        return(RADPUL_CUDA_MEM_COPY_HOST_DEVICE);
    }
    logMessage(debug, true, "CUDA host->device cos lookup table data transfer successful...\n");

    // transfer lookup table parameters to device
    float lutParam = ERP_TWO_PI;
    cuResult = hipModuleGetGlobal(&cudSymbol, NULL, cuModuleMain, "LUT_TWO_PI");
    if(cuResult != hipSuccess) {
            logMessage(error, true, "Couldn't get CUDA LTP symbol handle (error: %i)!\n", cuResult);
            return(RADPUL_CUDA_LOOKUP_SYMBOL);
    }
    cuResult = hipMemcpyHtoD(cudSymbol, &lutParam, sizeof(float));
    if(cuResult != hipSuccess)
    {
        logMessage(error, true, "Error during CUDA host->device lookup table parameter transfer (error: %d)\n", cuResult);
        return(RADPUL_CUDA_MEM_COPY_HOST_DEVICE);
    }
    logMessage(debug, true, "CUDA host->device lookup table parameter transfer successful...\n");

    lutParam = ERP_TWO_PI_INV;
    cuResult = hipModuleGetGlobal(&cudSymbol, NULL, cuModuleMain, "LUT_TWO_PI_INV");
    if(cuResult != hipSuccess) {
            logMessage(error, true, "Couldn't get CUDA LTPI symbol handle (error: %i)!\n", cuResult);
            return(RADPUL_CUDA_LOOKUP_SYMBOL);
    }
    cuResult = hipMemcpyHtoD(cudSymbol, &lutParam, sizeof(float));
    if(cuResult != hipSuccess)
    {
        logMessage(error, true, "Error during CUDA host->device lookup table parameter transfer (error: %d)\n", cuResult);
        return(RADPUL_CUDA_MEM_COPY_HOST_DEVICE);
    }
    logMessage(debug, true, "CUDA host->device lookup table parameter transfer successful...\n");

    // retrieve global symbol used later on
    cuResult = hipModuleGetGlobal(&timeSeriesLengthDeviceBuffer, NULL, cuModuleMain, "timeSeriesLength");
    if(cuResult != hipSuccess) {
            logMessage(error, true, "Couldn't get CUDA TSL symbol handle (error: %i)!\n", cuResult);
            return(RADPUL_CUDA_LOOKUP_SYMBOL);
    }

    // return allocated device pointer to caller

    output_dip->device_ptr=resampledTimeSeriesDeviceBuffer;

    return 0;
}


int run_resampling(DIfloatPtr input_dip, DIfloatPtr output_dip, const RESAMP_PARAMS *const params)
{
    // unused (doesn't prevent nvcc warnings, oh well)
    float * input = NULL;

    hipDeviceptr_t resampledTimeSeriesDeviceBuffer = output_dip.device_ptr;

    // kernel parameter offset counter (used per kernel launch, reset to 0 accordingly!)
    int kernelParamOffset = 0;

    // output variables
    int n_steps = 0;
    float mean = 0.0f;

    // compute time offsets

    dim3 dimBlockResampOffsets(CUDA_RESAMP_OFFSETS_BLOCKDIM_X);
    dim3 dimGridResampOffsets(params->nsamples_unpadded / dimBlockResampOffsets.x);

    logMessage(debug, true, "Executing time series modulation CUDA kernel (%u threads each in %u blocks)...\n", dimBlockResampOffsets.x, dimGridResampOffsets.x);

    // prepare parameters
    KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(modTimeOffsetsDeviceBuffer));
    cuResult = cuParamSetv(kernelTimeSeriesModulation, kernelParamOffset, &modTimeOffsetsDeviceBuffer, sizeof(modTimeOffsetsDeviceBuffer));
    kernelParamOffset += sizeof(modTimeOffsetsDeviceBuffer);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error during CUDA TSM kernel parameter setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(params->tau));
    cuResult = cuParamSetf(kernelTimeSeriesModulation, kernelParamOffset, params->tau);
    kernelParamOffset += sizeof(params->tau);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error during CUDA TSM kernel parameter setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(params->Omega));
    cuResult = cuParamSetf(kernelTimeSeriesModulation, kernelParamOffset, params->Omega);
    kernelParamOffset += sizeof(params->Omega);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error during CUDA TSM kernel parameter setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(params->Psi0));
    cuResult = cuParamSetf(kernelTimeSeriesModulation, kernelParamOffset, params->Psi0);
    kernelParamOffset += sizeof(params->Psi0);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error during CUDA TSM kernel parameter setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(params->dt));
    cuResult = cuParamSetf(kernelTimeSeriesModulation, kernelParamOffset, params->dt);
    kernelParamOffset += sizeof(params->dt);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error during CUDA TSM kernel parameter setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(params->step_inv));
    cuResult = cuParamSetf(kernelTimeSeriesModulation, kernelParamOffset, params->step_inv);
    kernelParamOffset += sizeof(params->step_inv);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error during CUDA TSM kernel parameter setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(params->S0));
    cuResult = cuParamSetf(kernelTimeSeriesModulation, kernelParamOffset, params->S0);
    kernelParamOffset += sizeof(params->S0);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error during CUDA TSM kernel parameter setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    cuResult = cuParamSetSize(kernelTimeSeriesModulation, kernelParamOffset);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error finalizing CUDA TSM kernel parameter setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    // prepare block size (CUDA_RESAMP_OFFSETS_BLOCKDIM_X threads per block (1D))
    cuResult = cuFuncSetBlockShape(kernelTimeSeriesModulation, dimBlockResampOffsets.x, dimBlockResampOffsets.y, dimBlockResampOffsets.z);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error during CUDA TSM kernel block setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    // launch kernel grid (n_unpadded/CUDA_RESAMP_OFFSETS_BLOCKDIM_X blocks in 1D grid)
    cuResult = cuLaunchGrid(kernelTimeSeriesModulation, dimGridResampOffsets.x, dimGridResampOffsets.y);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error launching CUDA TSM kernel (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_INVOKE);
    }

    // determine modulated time series length (done in kernel because global sync required and to avoid bus transfer of del_t as well as waste of host memory)

    dim3 dimBlockResampLength(1); // single thread per block (1D)
    dim3 dimGridResampLength(1);  // single block in grid (1D)

    logMessage(debug, true, "Executing modulated time series length CUDA kernel (%u threads each in %u blocks)...\n", dimBlockResampLength.x, dimGridResampLength.x);

    // prepare parameters
    kernelParamOffset = 0;
    KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(modTimeOffsetsDeviceBuffer));
    cuResult = cuParamSetv(kernelTimeSeriesLengthModulated, kernelParamOffset, &modTimeOffsetsDeviceBuffer, sizeof(modTimeOffsetsDeviceBuffer));
    kernelParamOffset += sizeof(modTimeOffsetsDeviceBuffer);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error during CUDA TSLM kernel parameter setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(params->nsamples_unpadded));
    cuResult = cuParamSeti(kernelTimeSeriesLengthModulated, kernelParamOffset, params->nsamples_unpadded);
    kernelParamOffset += sizeof(params->nsamples_unpadded);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error during CUDA TSLM kernel parameter setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    cuResult = cuParamSetSize(kernelTimeSeriesLengthModulated, kernelParamOffset);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error finalizing CUDA TSLM kernel parameter setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    // prepare block size (one single thread)
     cuResult = cuFuncSetBlockShape(kernelTimeSeriesLengthModulated, dimBlockResampLength.x, dimBlockResampLength.y, dimBlockResampLength.z);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error during CUDA TSLM kernel block setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    // launch kernel grid (one single block)
    cuResult = cuLaunchGrid(kernelTimeSeriesLengthModulated, dimGridResampLength.x, dimGridResampLength.y);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error launching CUDA TSLM kernel (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_INVOKE);
    }

    // return computed time series length
    cuResult = hipMemcpyDtoH(&n_steps, timeSeriesLengthDeviceBuffer, sizeof(int));
    if(cuResult != hipSuccess)
    {
        logMessage(error, true, "Error during CUDA device->host time series length transfer (error: %d)\n", cuResult);
        return(RADPUL_CUDA_MEM_COPY_DEVICE_HOST);
    }
    logMessage(debug, true, "CUDA device->host time series length (%d) transfer successful...\n", n_steps);

    // compute resampled time series (unpadded)

    dim3 dimBlockResamp(CUDA_RESAMP_BLOCKDIM_X);             // CUDA_RESAMP_BLOCKDIM_X threads per block (1D)
    dim3 dimGridResamp(params->nsamples / dimBlockResamp.x); // nsamples/CUDA_RESAMP_BLOCKDIM_X blocks in grid (1D)

    logMessage(debug, true, "Executing time series resampling CUDA kernel (%u threads each in %u blocks)...\n", dimBlockResamp.x, dimGridResamp.x);

    // prepare parameters
    kernelParamOffset = 0;
    KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(originalTimeSeriesDeviceBuffer));
    cuResult = cuParamSetv(kernelTimeSeriesResampling, kernelParamOffset, &originalTimeSeriesDeviceBuffer, sizeof(originalTimeSeriesDeviceBuffer));
    kernelParamOffset += sizeof(originalTimeSeriesDeviceBuffer);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error during CUDA TSR kernel parameter setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(modTimeOffsetsDeviceBuffer));
    cuResult = cuParamSetv(kernelTimeSeriesResampling, kernelParamOffset, &modTimeOffsetsDeviceBuffer, sizeof(modTimeOffsetsDeviceBuffer));
    kernelParamOffset += sizeof(modTimeOffsetsDeviceBuffer);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error during CUDA TSR kernel parameter setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(resampledTimeSeriesDeviceBuffer));
    cuResult = cuParamSetv(kernelTimeSeriesResampling, kernelParamOffset, &resampledTimeSeriesDeviceBuffer, sizeof(resampledTimeSeriesDeviceBuffer));
    kernelParamOffset += sizeof(resampledTimeSeriesDeviceBuffer);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error during CUDA TSR kernel parameter setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(timeSeriesMeanDeviceBuffer));
    cuResult = cuParamSetv(kernelTimeSeriesResampling, kernelParamOffset, &timeSeriesMeanDeviceBuffer, sizeof(timeSeriesMeanDeviceBuffer));
    kernelParamOffset += sizeof(timeSeriesMeanDeviceBuffer);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error during CUDA TSR kernel parameter setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(params->nsamples_unpadded));
    cuResult = cuParamSeti(kernelTimeSeriesResampling, kernelParamOffset, params->nsamples_unpadded);
    kernelParamOffset += sizeof(params->nsamples_unpadded);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error during CUDA TSR kernel parameter setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(n_steps));
    cuResult = cuParamSeti(kernelTimeSeriesResampling, kernelParamOffset, n_steps);
    kernelParamOffset += sizeof(n_steps);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error during CUDA TSR kernel parameter setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    cuResult = cuParamSetSize(kernelTimeSeriesResampling, kernelParamOffset);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error finalizing CUDA TSR kernel parameter setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    // prepare block size (CUDA_RESAMP_BLOCKDIM_X threads per block (1D))
     cuResult = cuFuncSetBlockShape(kernelTimeSeriesResampling, dimBlockResamp.x, dimBlockResamp.y, dimBlockResamp.z);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error during CUDA TSR kernel block setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    // launch kernel grid (params->nsamples_unpadded / CUDA_RESAMP_BLOCKDIM_X blocks in 1D grid)
    cuResult = cuLaunchGrid(kernelTimeSeriesResampling, dimGridResamp.x, dimGridResamp.y);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error launching CUDA TSR kernel (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_INVOKE);
    }

    // compute time series mean value

    // TODO: n_steps (effective length) threads would by sufficient
    int threadsPerBlockX = CUDA_RESAMP_REDUCTION_BLOCKDIM_X;  // CUDA_RESAMP_REDUCTION_BLOCKDIM_X threads per block (1D)

    // sum reduction loop control variables
    hipDeviceptr_t currentInputBuffer = NULL;
    hipDeviceptr_t currentOutputBuffer = NULL;
    int requiredBlocks = params->nsamples_unpadded / CUDA_RESAMP_REDUCTION_BLOCKDIM_X;
    int secondHalfOffset = requiredBlocks;
    bool useSecondHalfForOutput = false;
    int i = 1;

    do {
        // use kernel decomposition to do sum reduction (facilitates global memory sync, kernel invocations are cheap)
        logMessage(debug, true, "Executing time series mean reduction CUDA kernel (iteration %i using %i blocks of %u threads)...\n", i, requiredBlocks, threadsPerBlockX);

        // only the first iteration uses the resampled time series as input (obviously)
        if(i==1) {
            currentInputBuffer = resampledTimeSeriesDeviceBuffer;
        }
        else {
            // otherwise: alternate between first and second half of mean device buffer for input (inversely to output buffer)
            currentInputBuffer = (hipDeviceptr_t) ((float*)timeSeriesMeanDeviceBuffer + (useSecondHalfForOutput ? 0 : secondHalfOffset));
        }

        // alternate between first and second half of mean device buffer for output (inversely to input buffer)
        currentOutputBuffer = (hipDeviceptr_t) ((float*)timeSeriesMeanDeviceBuffer + (useSecondHalfForOutput ? secondHalfOffset : 0));

        // prepare parameters
        kernelParamOffset = 0;
        KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(currentInputBuffer));
        cuResult = cuParamSetv(kernelTimeSeriesMeanReduction, kernelParamOffset, &currentInputBuffer, sizeof(currentInputBuffer));
        kernelParamOffset += sizeof(currentInputBuffer);
        if(cuResult != hipSuccess) {
            logMessage(error, true, "Error during CUDA TSMR-%i kernel parameter setup (error: %d)\n", i, cuResult);
            return(RADPUL_CUDA_KERNEL_PREPARE);
        }

        KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(currentOutputBuffer));
        cuResult = cuParamSetv(kernelTimeSeriesMeanReduction, kernelParamOffset, &currentOutputBuffer, sizeof(currentOutputBuffer));
        kernelParamOffset += sizeof(currentOutputBuffer);
        if(cuResult != hipSuccess) {
            logMessage(error, true, "Error during CUDA TSMR-%i kernel parameter setup (error: %d)\n", i, cuResult);
            return(RADPUL_CUDA_KERNEL_PREPARE);
        }

        cuResult = cuParamSetSize(kernelTimeSeriesMeanReduction, kernelParamOffset);
        if(cuResult != hipSuccess) {
            logMessage(error, true, "Error finalizing CUDA TSMR-%i kernel parameter setup (error: %d)\n", i, cuResult);
            return(RADPUL_CUDA_KERNEL_PREPARE);
        }

        // prepare block size (CUDA_RESAMP_REDUCTION_BLOCKDIM_X)
        cuResult = cuFuncSetBlockShape(kernelTimeSeriesMeanReduction, threadsPerBlockX, 1, 1);
        if(cuResult != hipSuccess) {
            logMessage(error, true, "Error during CUDA TSMR-%i kernel block setup (error: %d)\n", i, cuResult);
            return(RADPUL_CUDA_KERNEL_PREPARE);
        }

        // launch kernel grid (iteratively reduced number of threads)
        cuResult = cuLaunchGrid(kernelTimeSeriesMeanReduction, requiredBlocks, 1);
        if(cuResult != hipSuccess) {
            logMessage(error, true, "Error launching CUDA TSMR-%i kernel (error: %d)\n", i, cuResult);
            return(RADPUL_CUDA_KERNEL_INVOKE);
        }

        // required blocks for next iteration
        if (requiredBlocks >= CUDA_RESAMP_REDUCTION_BLOCKDIM_X) {
            // we still can fill full blocks
            requiredBlocks /= CUDA_RESAMP_REDUCTION_BLOCKDIM_X;
        }
        else {
            if (requiredBlocks == 1) {
                // this was the final summing by the last block
                break;
            }
            else {
                // we're now within the last block (with fewer than blocksize elements), so sum pairs with one thread each
                threadsPerBlockX = requiredBlocks;
                requiredBlocks = 1;
            }
        }

        // flip output buffer specifier
        useSecondHalfForOutput = useSecondHalfForOutput ? false : true;

        // update progress counter
        i++;
    }
    while(requiredBlocks > 0);

    // return computed time series mean (first element of output buffer)
    cuResult = hipMemcpyDtoH(&mean, currentOutputBuffer, sizeof(float));
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error during CUDA device->host time series mean transfer (error: %i)\n", cuResult);
        return(RADPUL_CUDA_MEM_COPY_DEVICE_HOST);
    }
    logMessage(debug, true, "CUDA device->host time series sum (%f) transfer successful...\n", mean);

    // compute actual mean
    mean /= n_steps;

    logMessage(debug, true, "Actual time series mean is: %e\n", mean);

    // apply mean padding to time series

    // TODO: params->nsamples-n_steps threads would be sufficient
    dim3 dimBlockResampPadding(CUDA_RESAMP_PADDING_BLOCKDIM_X);            // CUDA_RESAMP_PADDING_BLOCKDIM_X threads per block (1D)
    dim3 dimGridResampPadding(params->nsamples / dimBlockResampPadding.x); // nsamples/CUDA_RESAMP_BLOCKDIM_X blocks in grid (1D)

    logMessage(debug, true, "Executing time series padding CUDA kernel (%u threads each in %u blocks)...\n", dimBlockResampPadding.x, dimGridResampPadding.x);

    // prepare parameters
    kernelParamOffset = 0;
    KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(resampledTimeSeriesDeviceBuffer));
    cuResult = cuParamSetv(kernelTimeSeriesPadding, kernelParamOffset, &resampledTimeSeriesDeviceBuffer, sizeof(resampledTimeSeriesDeviceBuffer));
    kernelParamOffset += sizeof(resampledTimeSeriesDeviceBuffer);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error during CUDA TSR kernel parameter setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(mean));
    cuResult = cuParamSetf(kernelTimeSeriesPadding, kernelParamOffset, mean);
    kernelParamOffset += sizeof(mean);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error during CUDA TSP kernel parameter setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(n_steps));
    cuResult = cuParamSeti(kernelTimeSeriesPadding, kernelParamOffset, n_steps);
    kernelParamOffset += sizeof(n_steps);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error during CUDA TSP kernel parameter setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    cuResult = cuParamSetSize(kernelTimeSeriesPadding, kernelParamOffset);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error finalizing CUDA TSP kernel parameter setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    // prepare block size (CUDA_RESAMP_PADDING_BLOCKDIM_X)
     cuResult = cuFuncSetBlockShape(kernelTimeSeriesPadding, dimBlockResampPadding.x, dimBlockResampPadding.y, dimBlockResampPadding.z);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error during CUDA TSP kernel block setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    // launch kernel grid (params->nsamples / CUDA_RESAMP_PADDING_BLOCKDIM_X blocks in 1D grid)
    cuResult = cuLaunchGrid(kernelTimeSeriesPadding, dimGridResampPadding.x, dimGridResampPadding.y);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error launching CUDA TSP kernel (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_INVOKE);
    }

    return 0;
}


int tear_down_resampling(DIfloatPtr output)
{
    hipDeviceptr_t resampledTimeSeriesDeviceBuffer = output.device_ptr;

    cuResult = hipFree(resampledTimeSeriesDeviceBuffer);
    if(cuResult != hipSuccess)
    {
        logMessage(error, true, "Error deallocating resampled time series device memory (error: %i)\n", cuResult);
        hipFree(timeSeriesMeanDeviceBuffer);
        return(RADPUL_CUDA_MEM_FREE_DEVICE);
    }

    cuResult = hipFree(originalTimeSeriesDeviceBuffer);
    if(cuResult != hipSuccess)
    {
        logMessage(error, true, "Error deallocating original time series device memory (error: %i)\n", cuResult);
        hipFree(modTimeOffsetsDeviceBuffer);
        hipFree(timeSeriesMeanDeviceBuffer);
        return(RADPUL_CUDA_MEM_FREE_DEVICE);
    }

    cuResult = hipFree(modTimeOffsetsDeviceBuffer);
    if(cuResult != hipSuccess)
    {
        logMessage(error, true, "Error deallocating modulated time offsets device memory (error: %i)\n", cuResult);
        hipFree(timeSeriesMeanDeviceBuffer);
        return(RADPUL_CUDA_MEM_FREE_DEVICE);
    }


    cuResult = hipFree(timeSeriesMeanDeviceBuffer);
    if(cuResult != hipSuccess)
    {
        logMessage(error, true, "Error deallocating time series mean reduction device memory (error: %i)\n", cuResult);
        return(RADPUL_CUDA_MEM_FREE_DEVICE);
    }

    return 0;
}


int set_up_fft(DIfloatPtr input_dip, DIfloatPtr *output_dip, uint32_t nsamples, unsigned int fft_size)
{

    // unused (doesn't prevent nvcc warnings, oh well)
    float * input = NULL;
    hipDeviceptr_t * output =&(output_dip->device_ptr); // powerspectrum on device memory

    // increase powerspectrum buffer length such that it matches the powerspectrum kernel's blocklength (no further control flow required in kernel)
    const unsigned int fft_size_padded = PADDED_FFT_SIZE(fft_size);

    logMessage(debug, true, "Padding output size of FFT with %u samples from %u to %u...\n", nsamples, fft_size,
fft_size_padded);

    // create fft plan
    cufResult = hipfftPlan1d(&cufPlan, nsamples, HIPFFT_R2C, 1);
    if(cufResult != HIPFFT_SUCCESS)
    {
        logMessage(error, true, "Error creating CUDA FFT plan (error code: %i)\n", cufResult);
        return(RADPUL_CUDA_FFT_PLAN);
    }
    logMessage(debug, true, "Created CUFFT plan...\n");

    // ensure FFTW compatibility
    cufResult = cufftSetCompatibilityMode(cufPlan, CUFFT_COMPATIBILITY_FFTW_ALL);
    if(cufResult != HIPFFT_SUCCESS)
    {
            logMessage(error, true, "Error setting CUDA FFTW compatibility (error code: %i)\n", cufResult);
            return(RADPUL_CUDA_FFT_PLAN);
    }


    // allocate device memory for power spectrum
    cuResult = hipMalloc(output, sizeof(float) * fft_size_padded);
    if(cuResult != hipSuccess)
    {
        logMessage(error, true, "Error allocating power spectrum device memory: %i bytes (error: %i)\n", sizeof(float) * fft_size_padded, cuResult);
        return(RADPUL_CUDA_MEM_ALLOC_DEVICE);
    }
    logMessage(debug, true, "Allocated power spectrum device memory: %i bytes\n", sizeof(float) * fft_size_padded);

    return 0;
}


int run_fft(DIfloatPtr input, DIfloatPtr output, uint32_t nsamples, unsigned int fft_size, float norm_factor)
{
    hipDeviceptr_t psDeviceBuffer=output.device_ptr;
    hipDeviceptr_t resampledTimeSeriesDeviceBuffer=input.device_ptr;

    // increase number of powerspectrum threads such that all blocks/threads are used completely (no further control flow required in kernel)
    static const unsigned int fft_size_padded = PADDED_FFT_SIZE(fft_size);

    // execute FFT
    cufResult = hipfftExecR2C(cufPlan, (hipfftReal*)resampledTimeSeriesDeviceBuffer, (hipfftComplex*)resampledTimeSeriesDeviceBuffer);
    if(cufResult != HIPFFT_SUCCESS)
    {
        logMessage(error, true, "Error executing CUDA FFT plan (error code: %i)\n", cufResult);
        hipfftDestroy(cufPlan);
        return(RADPUL_CUDA_FFT_EXEC);
    }
    logMessage(debug, true, "CUDA FFT execution successful...\n");

    // compute powerspectrum using CUDA kernel
    dim3 dimBlockFFT(CUDA_FFT_BLOCKDIM_X);            // CUDA_BLOCKDIM_X threads per block (1D)
    dim3 dimGridFFT(fft_size_padded / dimBlockFFT.x); // (fft_size_padded/CUDA_FFT_BLOCKDIM_X) blocks in grid (1D)

    logMessage(debug, true, "Executing power spectrum CUDA kernel (%u threads each in %u blocks)...\n", dimBlockFFT.x, dimGridFFT.x);

    // prepare parameters
    int kernelParamOffset = 0;

    KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(resampledTimeSeriesDeviceBuffer));
    cuResult = cuParamSetv(kernelPowerSpectrum, kernelParamOffset, &resampledTimeSeriesDeviceBuffer, sizeof(resampledTimeSeriesDeviceBuffer));
    kernelParamOffset += sizeof(resampledTimeSeriesDeviceBuffer);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error during CUDA PS kernel parameter setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(psDeviceBuffer));
    cuResult = cuParamSetv(kernelPowerSpectrum, kernelParamOffset, &psDeviceBuffer, sizeof(psDeviceBuffer));
    kernelParamOffset += sizeof(psDeviceBuffer);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error during CUDA PS kernel parameter setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    KERNEL_PARAM_ALIGN_UP(kernelParamOffset, __alignof(norm_factor));
    cuResult = cuParamSetf(kernelPowerSpectrum, kernelParamOffset, norm_factor);
    kernelParamOffset += sizeof(norm_factor);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error during CUDA PS kernel parameter setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    cuResult = cuParamSetSize(kernelPowerSpectrum, kernelParamOffset);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error finalizing CUDA PS kernel parameter setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    // prepare block size (CUDA_FFT_BLOCKDIM_X)
     cuResult = cuFuncSetBlockShape(kernelPowerSpectrum, dimBlockFFT.x, dimBlockFFT.y, dimBlockFFT.z);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error during CUDA PS kernel block setup (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_PREPARE);
    }

    // launch kernel grid
    cuResult = cuLaunchGrid(kernelPowerSpectrum, dimGridFFT.x, dimGridFFT.y);
    if(cuResult != hipSuccess) {
        logMessage(error, true, "Error launching CUDA PS kernel (error: %d)\n", cuResult);
        return(RADPUL_CUDA_KERNEL_INVOKE);
    }

    return 0;
}


int tear_down_fft(DIfloatPtr output_dip)
{
    hipDeviceptr_t psDeviceBuffer = output_dip.device_ptr;

    cuResult = hipFree(psDeviceBuffer);
    if(cuResult != hipSuccess)
    {
        logMessage(error, true, "Error deallocating power spectrum device memory (error: %i)\n", cuResult);
        hipfftDestroy(cufPlan);
        return(RADPUL_CUDA_MEM_FREE_DEVICE);
    }

    cufResult = hipfftDestroy(cufPlan);
    if(cufResult != HIPFFT_SUCCESS)
    {
        logMessage(error, true, "Error destroying CUDA FFT plan (error code: %i)\n", cufResult);
        return(RADPUL_CUDA_FFT_DESTROY);
    }

    return 0;
}


int shutdown_cuda()
{
    // destroy context
    cuResult = hipCtxDestroy(cuContext);
    if(cuResult != hipSuccess) {
        logMessage(warn, true, "Couldn't destroy CUDA context (error: %i)!\n", cuResult);
    }

    logMessage(debug, true, "CUDA shutdown successful...\n");

    return 0;
}
